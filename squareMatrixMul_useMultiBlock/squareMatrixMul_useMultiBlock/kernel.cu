#include "hip/hip_runtime.h"
//GPU�������� ó���غ��� ����
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <time.h> //clock(), time_tŸ���� ����

__global__ void squareMatrixMulKernel(int *c, int *a, int *b, int arrayWidth);	//�ٷ� ���� ������ �Լ� �ȿ��� ������ ������ �Բ� ȣ��Ǵ� �Լ�, host���� ȣ�Ⱑ���ϸ� Device���� ����Ǵ� Ŀ�� �Լ�
hipError_t squareMatrixMulWithGPU(int *c, int *a, int *b, int arrayWidth);		// �� ��������� ���������� GPU���� �����ϴ� �Լ�
void squareMatrixMulWithCPU(int *c, int *a, int *b, int arrayWidth);			// �� ��������� ���������� CPU���� �����ϴ� �Լ� 
void initArrayToRandom(int *array, int arrayWidth);								// ������ ���� ����� �ʱ�ȭ �ϴ� �Լ� 
void initArrayToZero(int *array, int arrayWidth);								// 0���� ����� �ʱ�ȭ �ϴ� �Լ�
void printArrayAllElement(int *array, int arrayWidth);							// ����� ��� ���Ҹ� ����ϴ� �Լ�

int main()
{
	int *a;
	int *b;
	int *c;

	int arrayWidth;

	//�� �Լ��� �����Լ����� �۵��ؾ� ���� ���� �ȳ����Ե�.
	srand(time(NULL));

	while (1)
	{
		printf("��ĳʺ� : ");
		scanf("%d", &arrayWidth);

		a = (int*)malloc(sizeof(int)*arrayWidth*arrayWidth);
		b = (int*)malloc(sizeof(int)*arrayWidth*arrayWidth);
		c = (int*)malloc(sizeof(int)*arrayWidth*arrayWidth);


		//������ �����ϱ��� ������ �ʱ�ȭ
		initArrayToRandom(a, arrayWidth);
		initArrayToRandom(b, arrayWidth);

		//�� ��������� ��������(CPU����)
		initArrayToZero(c, arrayWidth);
		squareMatrixMulWithCPU(c, a, b, arrayWidth);


		/*
		printArrayAllElement(a, arrayWidth);
		printf("\n");
		printArrayAllElement(b, arrayWidth);
		printf("\n");
		*/

		//printArrayAllElement(c, arrayWidth);
		//printf("\n");


		//�� ��������� ��������(GPU����)
		initArrayToZero(c, arrayWidth);
		hipError_t cudaStatus = squareMatrixMulWithGPU(c, a, b, arrayWidth);


		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "squareMatrixMulWithGPU failed!");
			return 1;
		}

		//���Ȯ��
		//printArrayAllElement(c, arrayWidth);

		// hipDeviceReset must be called before exiting in order for profiling and
		// tracing tools such as Nsight and Visual Profiler to show complete traces.
		cudaStatus = hipDeviceReset();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipDeviceReset failed!");
			return 1;
		}

		free(a);
		free(b);
		free(c);
	}


	return 0;
}

__global__ void squareMatrixMulKernel(int *c, int *a, int *b, int arrayWidth)
{
	float sum = 0;

	//��Ŀ��� ����Ϸ��� �ϴ� ��ġ�� �ε��� �̰��� ����ȭ �Ȱ��̹Ƿ� �ܿ��� ����.
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int row = blockIdx.y * blockDim.y + threadIdx.y;


	//��ϴ� �����尡 4x4�̰�
	//����� ������ 1x1�̸�
	//printf("%d, %d / %d, %d / %d, %d\n", blockDim.x, blockDim.y, blockIdx.x, blockIdx.y, threadIdx.x, threadIdx.y);
	// 4, 4, 0, 0, x, y �̷��� �տ� 4���� ���ڴ� ������ ���� �� �� �־���.
	//blockDim : ��� ���ʿ� ���Ե� �����尡 � ��x�� �������� �Ǿ��ִ���.
	//blockIdx : ����� �ε���
	//threadIdx : �������� �ε���

	for (int i = 0; i < arrayWidth; ++i)
	{
		float Aelement = a[row * arrayWidth + i];
		float Belement = b[i*arrayWidth + col];
		sum += Aelement * Belement;
	}
	c[row * arrayWidth + col] = sum;
}

hipError_t squareMatrixMulWithGPU(int *c, int *a, int *b, int arrayWidth)
{
	int *dev_a = 0;
	int *dev_b = 0;
	int *dev_c = 0;

	hipError_t cudaStatus;




	//������ ��� �ϳ��� �������� ���� 16x16�̶�� ����.
	//�׷��� 16���� �����鼭 arrayWidth�� ������� ���� ã�Ƽ� �� ���� ��� �ϳ��� �������� ���� ��������� �Ѵ�. 
	int threadWidth = 0;

	for (int i = 16; i > 0; --i)
	{
		if (arrayWidth % i == 0)
		{
			threadWidth = i;
			break;
		}
	}

	//���⼭ dimBlock�� ũ��� �Է¹��� �迭�� �ʺ� ���� �޶����� �Ѵ�. �ȱ׷��� ������ Ʋ���� �߸��� ����� �޾ƺ� �� ���� ���̴�. 
	dim3 dimBlock(threadWidth, threadWidth);												// ��� �ϳ� �� ������ �� 
	dim3 dimGrid(arrayWidth / dimBlock.x, arrayWidth / dimBlock.y);		// ������ ����� ����

																		//����) �ʺ� 12�� ��� �� 12*12����� �����Ѵ� ġ��. �׷��� �츮�� ��� �ϳ��� 4*4�� �����带 ������ �Ͽ����Ƿ� �ѹ��� ����ó���� �����ع������� ����� 3*3���� ����� �ʿ��ϴ�.
	printf("��ϴ� ������ �� : %d x %d, ����� �� : %d x %d\n\n", threadWidth, threadWidth, arrayWidth / dimBlock.x, arrayWidth / dimBlock.y);



	//��Ƽ GPU �ý��� ȯ�濡�� ������ GPU�� �����ϴ� �ڵ�. 
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	// Device(GPU)�� grid�� �ִ� Global Memory�� 3���� ���͸� ���� GPU���۸� �Ҵ��Ѵ�.
	cudaStatus = hipMalloc((void**)&dev_c, arrayWidth * arrayWidth * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_a, arrayWidth * arrayWidth * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_b, arrayWidth * arrayWidth * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	// ȣ��Ʈ�� �����ϴ� ���۸� Device(GPU)�� �����ϴ� GPU���۵�� �����Ѵ�.
	cudaStatus = hipMemcpy(dev_a, a, arrayWidth * arrayWidth * sizeof(int), hipMemcpyHostToDevice); //hipMemcpy�� �񵿱��������� �۵��ϸ� HostToHost, HostToDevice, DeviceToHost, DeviceToDevice  4���� Ÿ���� ������.
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_b, b, arrayWidth * arrayWidth * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}


	hipEvent_t start, stop;
	float gapTime = 0;

	//�̺�Ʈ ��ü����
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start, 0);		 //���۽ð� ����


									 // �� ��ҿ� ���� �ϳ��� ������� GPU���� Ŀ�� ����
									 // N���� thread���� �� data pair�� ���Ͽ� �� ������ addKernel( )�� ����
	squareMatrixMulKernel << < dimGrid, dimBlock >> > (dev_c, dev_a, dev_b, arrayWidth); // ���� 2���� ������ �����带 �����ϴ� ����, ������ 4���� �������� Ŀ���Լ��� �Ű�����

	hipEventRecord(stop, 0);		//�����ð� ����

	hipEventSynchronize(stop); 	//stop�̺�Ʈ�� ��ϵ� �� ���� ���⼭ �����ִ´�.


	hipEventElapsedTime(&gapTime, start, stop); // ���۽ð��� ���� �ð��� ���� ����Ͽ� �����Ѵ�.

												 //�̺�Ʈ ��ü����
	hipEventDestroy(start);
	hipEventDestroy(stop);


	printf("����ð� ����(GPU) : %f ms\n", gapTime);

	// Ŀ���� �����ϴ� ���� ������ �־����� Ȯ���Ѵ�.
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "squareMatrixMulKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}


	// hipDeviceSynchronize�� Ŀ���� ����ĥ �� ���� ��ٸ���. �׸��� �� ���ൿ�ȿ� �߻��ߴ� ��� ������ ��ȯ�Ѵ�.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching squareMatrixMulKernel!\n", cudaStatus);
		goto Error;
	}


	// GPU buffer���� ȣ��Ʈ �޸𸮷� ������͸� �����Ѵ�.
	cudaStatus = hipMemcpy(c, dev_c, arrayWidth * arrayWidth * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

Error:
	hipFree(dev_c);
	hipFree(dev_a);
	hipFree(dev_b);

	return cudaStatus;
}

void squareMatrixMulWithCPU(int *c, int *a, int *b, int arrayWidth)
{
	clock_t startTime, endTime;
	double gapTime;

	startTime = clock(); //������ ���۽ð� üũ


	for (int i = 0; i < arrayWidth; i++) {
		for (int j = 0; j < arrayWidth; j++) {
			int sum = 0;

			for (int k = 0; k < arrayWidth; k++) {
				int hos_a = a[i*arrayWidth + k];
				int hos_b = b[k*arrayWidth + j];
				//printf("%d * %d\n", i*arrayWidth + k, k*arrayWidth + j);
				sum += hos_a*hos_b;
			}
			c[i*arrayWidth + j] = sum;
		}
	}

	endTime = clock();	//������ �����ð� üũ

	gapTime = (double)endTime - startTime; // �����Ŭ����ƽ�� / �ʴ�Ŭ����

	printf("����ð� ����(CPU) : %f ms\n", gapTime);
}

void initArrayToRandom(int *array, int arrayWidth)
{
	int arrayTotalCount = arrayWidth * arrayWidth;

	for (int i = 0; i < arrayTotalCount; i++)
		array[i] = rand() % 2; // 0 ~ 4-1 ������ �������� ����.
}

void initArrayToZero(int *array, int arrayWidth)
{
	int arrayTotalCount = arrayWidth * arrayWidth;

	for (int i = 0; i < arrayTotalCount; i++)
		array[i] = 0;
}

void printArrayAllElement(int *array, int arrayWidth)
{
	int arrayTotalCount = arrayWidth * arrayWidth;

	for (int i = 0; i < arrayTotalCount; i++)
	{
		printf("%2d ", array[i]);
		if ((i + 1) % arrayWidth == 0)
			printf("\n");
	}
}