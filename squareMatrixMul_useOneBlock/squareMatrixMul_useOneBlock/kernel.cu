#include "hip/hip_runtime.h"
//GPU�������� ó���غ��� ����
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <time.h> //clock(), time_tŸ���� ����

__global__ void squareMatrixMulKernel(int *c, int *a, int *b, int arrayWidth);	//�ٷ� ���� ������ �Լ� �ȿ��� ������ ������ �Բ� ȣ��Ǵ� �Լ�, host���� ȣ�Ⱑ���ϸ� Device���� ����Ǵ� Ŀ�� �Լ�
hipError_t squareMatrixMulWithGPU(int *c, int *a, int *b, int arrayWidth);		// �� ��������� ���������� GPU���� �����ϴ� �Լ�
void squareMatrixMulWithCPU(int *c, int *a, int *b, int arrayWidth);			// �� ��������� ���������� CPU���� �����ϴ� �Լ� 
void initArrayToRandom(int *array, int arrayWidth);								// ������ ���� ����� �ʱ�ȭ �ϴ� �Լ� 
void initArrayToZero(int *array, int arrayWidth);								// 0���� ����� �ʱ�ȭ �ϴ� �Լ�
void printArrayAllElement(int *array, int arrayWidth);							// ����� ��� ���Ҹ� ����ϴ� �Լ�

int main()
{
	const int arrayWidth = 16;	//��� �Ѱ����� �̿��� �����Ͽ����Ƿ� �� ��Ͽ� �ִ� 1024���� �����常 ��밡���ϰ� ���� 32�� �ִ� ���ġ�̴�. �̸� �Ѿ�� GPU���� ������ ���� ���̴�.

								//�Ʒ��� �ּ�ó���� ���� ������ε� ���� ����
	int a[arrayWidth*arrayWidth] = { 0 };
	int b[arrayWidth*arrayWidth] = { 0 };
	int c[arrayWidth*arrayWidth] = { 0 };

	/*
	int *a;
	int *b;
	int *c;

	a = (int*)malloc(sizeof(int)*arrayWidth*arrayWidth);
	b = (int*)malloc(sizeof(int)*arrayWidth*arrayWidth);
	c = (int*)malloc(sizeof(int)*arrayWidth*arrayWidth);
	*/

	//�� �Լ��� �����Լ����� �۵��ؾ� ���� ���� �ȳ����Ե�.
	srand(time(NULL));

	//������ �����ϱ��� ������ �ʱ�ȭ
	initArrayToRandom(a, arrayWidth);
	initArrayToRandom(b, arrayWidth);

	//�� ��������� ��������(CPU����)
	initArrayToZero(c, arrayWidth);
	squareMatrixMulWithCPU(c, a, b, arrayWidth);


	/*
	printArrayAllElement(a, arrayWidth);
	printf("\n");
	printArrayAllElement(b, arrayWidth);
	printf("\n");
	*/

	//printArrayAllElement(c, arrayWidth);
	//printf("\n");


	//�� ��������� ��������(GPU����)
	initArrayToZero(c, arrayWidth);
	hipError_t cudaStatus = squareMatrixMulWithGPU(c, a, b, arrayWidth);


	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "squareMatrixMulWithGPU failed!");
		return 1;
	}

	//���Ȯ��
	//printArrayAllElement(c, arrayWidth);

	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}

	return 0;
}

__global__ void squareMatrixMulKernel(int *c, int *a, int *b, int arrayWidth)
{
	float sum = 0;

	//���⼭ threadIdx.x�� y�� ����� �ε����� ����. ����) 2x2����϶� 00 01 10 11

	for (int i = 0; i < arrayWidth; ++i)
	{
		float Aelement = a[threadIdx.y * arrayWidth + i];
		float Belement = b[i*arrayWidth + threadIdx.x];
		sum += Aelement * Belement;
	}
	c[threadIdx.y * arrayWidth + threadIdx.x] = sum;
}

hipError_t squareMatrixMulWithGPU(int *c, int *a, int *b, int arrayWidth)
{
	int *dev_a = 0;
	int *dev_b = 0;
	int *dev_c = 0;

	hipError_t cudaStatus;


	dim3 dimGrid(1, 1);							// blocks per grid
	dim3 dimBlock(arrayWidth, arrayWidth);		// Threads per block


												//��Ƽ GPU �ý��� ȯ�濡�� ������ GPU�� �����ϴ� �ڵ�. 
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	// Device(GPU)�� grid�� �ִ� Global Memory�� 3���� ���͸� ���� GPU���۸� �Ҵ��Ѵ�.
	cudaStatus = hipMalloc((void**)&dev_c, arrayWidth * arrayWidth * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_a, arrayWidth * arrayWidth * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_b, arrayWidth * arrayWidth * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	// ȣ��Ʈ�� �����ϴ� ���۸� Device(GPU)�� �����ϴ� GPU���۵�� �����Ѵ�.
	cudaStatus = hipMemcpy(dev_a, a, arrayWidth * arrayWidth * sizeof(int), hipMemcpyHostToDevice); //hipMemcpy�� �񵿱��������� �۵��ϸ� HostToHost, HostToDevice, DeviceToHost, DeviceToDevice  4���� Ÿ���� ������.
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_b, b, arrayWidth * arrayWidth * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}


	hipEvent_t start, stop;
	float gapTime = 0;

	//�̺�Ʈ ��ü����
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start, 0);		 //���۽ð� ����


	squareMatrixMulKernel << < dimGrid, dimBlock >> > (dev_c, dev_a, dev_b, arrayWidth); // ���� 2���� ������ �����带 �����ϴ� ����, ������ 4���� �������� Ŀ���Լ��� �Ű�����

	hipEventRecord(stop, 0);		//�����ð� ����

	hipEventSynchronize(stop); 	//stop�̺�Ʈ�� ��ϵ� �� ���� ���⼭ �����ִ´�.


	hipEventElapsedTime(&gapTime, start, stop); // ���۽ð��� ���� �ð��� ���� ����Ͽ� �����Ѵ�.

												 //�̺�Ʈ ��ü����
	hipEventDestroy(start);
	hipEventDestroy(stop);


	printf("����ð� ����(GPU) : %f ms\n", gapTime);

	// Ŀ���� �����ϴ� ���� ������ �־����� Ȯ���Ѵ�.
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "squareMatrixMulKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}


	// hipDeviceSynchronize�� Ŀ���� ����ĥ �� ���� ��ٸ���. �׸��� �� ���ൿ�ȿ� �߻��ߴ� ��� ������ ��ȯ�Ѵ�.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching squareMatrixMulKernel!\n", cudaStatus);
		goto Error;
	}


	// GPU buffer���� ȣ��Ʈ �޸𸮷� ������͸� �����Ѵ�.
	cudaStatus = hipMemcpy(c, dev_c, arrayWidth * arrayWidth * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

Error:
	hipFree(dev_c);
	hipFree(dev_a);
	hipFree(dev_b);

	return cudaStatus;
}

void squareMatrixMulWithCPU(int *c, int *a, int *b, int arrayWidth)
{
	clock_t startTime, endTime;
	double gapTime;

	startTime = clock(); //������ ���۽ð� üũ


	for (int i = 0; i < arrayWidth; i++) {
		for (int j = 0; j < arrayWidth; j++) {
			int sum = 0;

			for (int k = 0; k < arrayWidth; k++) {
				int hos_a = a[i*arrayWidth + k];
				int hos_b = b[k*arrayWidth + j];
				//printf("%d * %d\n", i*arrayWidth + k, k*arrayWidth + j);
				sum += hos_a*hos_b;
			}
			c[i*arrayWidth + j] = sum;
		}
	}

	endTime = clock();	//������ �����ð� üũ

	gapTime = (double)endTime - startTime; // �����Ŭ����ƽ�� / �ʴ�Ŭ����

	printf("����ð� ����(CPU) : %f ms\n", gapTime);
}

void initArrayToRandom(int *array, int arrayWidth)
{
	int arrayTotalCount = arrayWidth * arrayWidth;

	for (int i = 0; i < arrayTotalCount; i++)
		array[i] = rand() % 2; // 0 ~ 4-1 ������ �������� ����.
}

void initArrayToZero(int *array, int arrayWidth)
{
	int arrayTotalCount = arrayWidth * arrayWidth;

	for (int i = 0; i < arrayTotalCount; i++)
		array[i] = 0;
}

void printArrayAllElement(int *array, int arrayWidth)
{
	int arrayTotalCount = arrayWidth * arrayWidth;

	for (int i = 0; i < arrayTotalCount; i++)
	{
		printf("%2d ", array[i]);
		if ((i + 1) % arrayWidth == 0)
			printf("\n");
	}
}